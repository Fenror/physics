#include "pbd_framework.h"
#include "oskgl.h"
#include "pbd_kernels.h"
#include <cuda_gl_interop.h>
#include <>

PBDFramework::PBDFramework()
{
  CreateVertexBuffers();
  InitializeArrays();
}

void PBDFramework::CreateVertexBuffers()
{

  /* Construct grid vertices */
  glm::vec3 grid_data[grid_size_][grid_size_];
  for (int i = 0; i < grid_size_; ++i)
  {
    for (int j = 0; j < grid_size_; ++j)
    {
      const float x = static_cast<float>(j)/(grid_size_-1);
      const float z = static_cast<float>(i)/(grid_size_-1);
      grid_data[i][j] = {x, 0.0f, z};
    }
  }

  /* Construct index buffer */
  glm::uvec3 grid_elements[grid_size_-1][grid_size_-1][2];
  for (int i = 0; i < grid_size_-1; ++i)
  {
    for (int j = 0; j < grid_size_-1; ++j)
    {
      grid_elements[i][j][0] =
        {i*grid_size_+j, (i+1)*grid_size_+j, (i+1)*grid_size_+j+1};
      grid_elements[i][j][1] =
        {i*grid_size_+j, (i+1)*grid_size_+j+1, i*grid_size_+j+1};
    }
  }

  /* Buffer data */
  glGenBuffers(1, &vertex_vbo_);
  glBindBuffer(GL_ARRAY_BUFFER, vertex_vbo_);
  glBufferData(GL_ARRAY_BUFFER, sizeof(grid_data), grid_data, GL_DYNAMIC_DRAW);
  glBindBuffer(GL_ARRAY_BUFFER, 0);

  glGenBuffers(1, &ibo);
  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, ibo);
  glBufferData(GL_ELEMENT_ARRAY_BUFFER, sizeof(grid_elements), grid_elements, GL_STATIC_READ);
  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

  /* Construct VAOs */
  glGenVertexArrays(1, &vertex_vao_);
  glBindVertexArray(vertex_vao_);
  glBindBuffer(GL_ARRAY_BUFFER, vertex_vbo_);
  glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, 0, 0);
  glEnableVertexAttribArray(0);
  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, ibo);

  glBindVertexArray(0);
  glBindBuffer(GL_ARRAY_BUFFER, 0);
  glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);

  hipGraphicsGLRegisterBuffer(&vertices_cuda_,
                               vertex_vbo_,
                               cudaGraphicsMapFlagsWriteDiscard);

  shader_ = oskgl::CompileShaderProgram("cloth.vsh", "cloth.fsh");
}

void PBDFramework::InitializeArrays()
{
  hipMalloc(&p_, num_vertices_*sizeof(glm::vec3));
  hipMalloc(&v_, num_vertices_*sizeof(glm::vec3));
  hipMemset(v_, 0, num_vertices_*sizeof(glm::vec3));
  hipMalloc(&ext_f_, num_vertices_*sizeof(glm::vec3));
  hipMemset(ext_f_, 0, num_vertices_*sizeof(glm::vec3));
}

void PBDFramework::Draw()
{
  glUseProgram(shader_);
  glBindVertexArray(vertex_vao_);
  glDrawElements(GL_TRIANGLES, 3*num_triangles_, GL_UNSIGNED_INT, 0);
  glBindVertexArray(0);
  glUseProgram(0);
}

void PBDFramework::Move()
{
  hipGraphicsMapResources(1, &vertices_cuda_, 0);
  size_t num_bytes;
  hipGraphicsResourceGetMappedPointer((void**) &x_,
                                       &num_bytes,
                                       vertices_cuda_);
  GuessNewPositions<<<grid_size_, grid_size_>>>(x_,
                                                v_,
                                                ext_f_,
                                                timestep_,
                                                inverse_vertex_mass_,
                                                p_);
  CopyPositions<<<grid_size_, grid_size_>>>(x_, p_);
  SetZero<<<grid_size_, grid_size_>>>(ext_f_);

  hipGraphicsUnmapResources(1, &vertices_cuda_, 0);
}

PBDFramework::~PBDFramework()
{
  hipFree(p_);
  hipFree(v_);
  hipFree(ext_f_);
}
