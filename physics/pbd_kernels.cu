#include "hip/hip_runtime.h"
#include <glm/glm.hpp>

__global__ void GuessNewPositions(glm::vec3* x,
                                  glm::vec3* v,
                                  glm::vec3* f,
                                  const float dt,
                                  const float w,
                                  glm::vec3* p)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  v[i] += dt*w*f[i];
  p[i] = x[i] + dt*v[i];
}

__global__ void UpdateVelocities(glm::vec3* x,
                                 glm::vec3* p,
                                 glm::vec3* v,
                                 const float dt)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  v[i] += (p[i] - x[i])/dt;
}

__global__ void ProjectLengthConstraints(glm::vec3* p)
{
}

__global__ void SetElement(glm::vec3* array, const int i, const glm::vec3 value)
{
  array[i] = value;
}

__global__ void SetZero(glm::vec3* array)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  array[i] = {0.0f, 0.0f, 0.0f};
}

__global__ void CopyPositions(glm::vec3* x,
                              glm::vec3* p)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  x[i] = p[i];
}
