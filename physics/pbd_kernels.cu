#include "hip/hip_runtime.h"
#include "pbd_kernels.h"

__device__ int grid_size = 10;

__device__ void ProjectLengthConstraint(glm::vec3* p, int i, int j, float d)
{
  glm::vec3 p1 = p[i];
  glm::vec3 p2 = p[j];
  p[i] -= 0.5f*(glm::length(p1 - p2) - d)*glm::normalize(p1-p2);
  p[j] += 0.5f*(glm::length(p1 - p2) - d)*glm::normalize(p1-p2);
}

__global__ void GuessNewPositions(glm::vec3* x,
                                  glm::vec3* v,
                                  glm::vec3* f,
                                  const float dt,
                                  const float w,
                                  glm::vec3* p)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  v[i] += dt*w*f[i];
  p[i] = x[i] + dt*v[i];
}

__global__ void UpdateVelocities(glm::vec3* x,
                                 glm::vec3* p,
                                 glm::vec3* v,
                                 const float dt)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  v[i] += (p[i] - x[i])/dt;
}

__global__ void ProjectLengthConstraints(glm::vec3* p)
{
  const float side_length = 1.0f/(grid_size-1);
  const float diag_length = glm::sqrt(2*side_length*side_length);
  if (blockDim.x & 1)
  {
    int i = (gridDim.x+1)*threadIdx.x + blockIdx.x;
    int j = i+1;
    ProjectLengthConstraint(p, i, j, side_length);
  }
  __syncthreads();
  if (!(blockDim.x & 1))
  {
    int i = (gridDim.x+1)*threadIdx.x + blockIdx.x;
    int j = i+1;
    ProjectLengthConstraint(p, i, j, side_length);
  }
  __syncthreads();
  if (blockDim.x & 1)
  {
    int i = (gridDim.x+1)*blockIdx.x + threadIdx.x;
    int j = i+gridDim.x+1;
    ProjectLengthConstraint(p, i, j, side_length);
  }
  __syncthreads();
  if (!(blockDim.x & 1))
  {
    int i = (gridDim.x+1)*blockIdx.x + threadIdx.x;
    int j = i+gridDim.x+1;
    ProjectLengthConstraint(p, i, j, side_length);
  }
  //__syncthreads();
  //if (threadIdx.x < grid_size-1)
  //{
  //  int i = (gridDim.x+1)*threadIdx.x + blockIdx.x;
  //  int j = i+gridDim.x+2;
  //  ProjectLengthConstraint(p, i, j, diag_length);
  //}
  //__syncthreads();
  //if (threadIdx.x < grid_size-1)
  //{
  //  int i = (gridDim.x+1)*threadIdx.x + blockIdx.x + 1;
  //  int j = i+gridDim.x;
  //  ProjectLengthConstraint(p, i, j, diag_length);
  //}
}

__global__ void SetElement(glm::vec3* array, const int i, const glm::vec3 value)
{
  array[i] = value;
}

__global__ void SetZero(glm::vec3* array)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  array[i] = {0.0f, 0.0f, 0.0f};
}

__global__ void CopyPositions(glm::vec3* x,
                              glm::vec3* p)
{
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  x[i] = p[i];
}
